#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>
#include "common.h"


__host__ static void
matrix_print(int32_t* mtx, uint32_t len, const char* label)
{
	printf("\n%s\n", label);
	for (uint32_t row = 0; row < len; ++row) {
		printf("\t\t");
		for (uint32_t col = 0; col < len; ++col) {
			printf("%02d ", mtx[row * len + col]);
		}
		printf("\n");
	}
}

__host__ static void
matrix_set_random(int32_t* mtx, uint32_t len)
{
	for (uint32_t row = 0; row < len; ++row)
		for (uint32_t col = 0; col < len; ++col) {
			mtx[row * len + col] = rand() % 10;
		}
}

__host__ static void
hostMalloc(int32_t** mtx, uint32_t len)
{
	*mtx = (int32_t*) malloc(len);

	if (NULL == *mtx) {
		fprintf(stderr, "Failed to allocate memory.\n"
						"file: %s\nline: %d\n",
						__FILE__, __LINE__);
		exit(1);
	}
}

__global__ void
kern_matrix_multiply(int32_t* first, int32_t* second, int32_t* output, uint32_t width)
{
	uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t value = 0;

	if (row >= width || col >= width)
		return;
	
	for (uint32_t k = 0; k < width; ++k)
		value += first[row * width + k] * second[k * width + col];
	
	output[row * width + col] = value;
}

static void
host_matrix_multiply(int32_t* first, int32_t* second, int32_t* output, uint32_t width)
{
	int32_t value = 0;
	for (uint32_t row = 0; row < width; ++row) {

		for (uint32_t col = 0; col < width; ++col) {
			for (uint32_t k = 0; k < width; ++k){
				value += first[row * width + k] * second[k * width + col];
			}
			output[row * width + col] += value;
		}
	}
}

int
main(int argc, char* argv[])
{
	uint32_t width = 2;
	int32_t* first = NULL;
	int32_t* second = NULL;
	int32_t* output = NULL;

	int32_t* dev_first = NULL;
	int32_t* dev_second = NULL;
	int32_t* dev_output = NULL;

	struct timeval start, stop;
	double elapsed;

	if (argc > 1) {
		width = strtoull(argv[1], NULL, 10);
	}

	/* Allocating memory */
	hostMalloc(&first, width);
	hostMalloc(&second, width);
	hostMalloc(&output, width);

	hipMalloc(&dev_first,  width * sizeof(int32_t));
	CUDA_ASSERT_ERROR();
	hipMalloc(&dev_second, width * sizeof(int32_t));
	CUDA_ASSERT_ERROR();
	hipMalloc(&dev_output, width * sizeof(int32_t));
	CUDA_ASSERT_ERROR();

	/* Initialization. */
	srand(time(NULL));
	matrix_set_random(first, width);
	matrix_set_random(second, width);
	
	hipMemcpy(dev_first, first, width, hipMemcpyHostToDevice);
	CUDA_ASSERT_ERROR();
	hipMemcpy(dev_second, second, width, hipMemcpyHostToDevice);
	CUDA_ASSERT_ERROR();

	dim3 grid_(ceil(width / (double)32), ceil(width / (double)32), 1);
	dim3 block_(32, 32, 1);

	/* Start point. */
	gettimeofday(&start, NULL);

	host_matrix_multiply(dev_first, dev_second, dev_output, width);
	kern_matrix_multiply<<<grid_, block_>>>(dev_first, dev_second, dev_output, width);
	CUDA_ASSERT_ERROR();

	/* logging result. */
	gettimeofday(&stop, NULL);

	elapsed = GET_MS(start, stop);
	printf("\nelapsed time: %.02f ms.\n", elapsed);
	
	hipMemcpy(output, dev_output, width, hipMemcpyDeviceToHost);
	CUDA_ASSERT_ERROR();

	matrix_print(first, width, "matrix A:");
	matrix_print(second, width, "matrix B:");
	matrix_print(output, width, "matrix C:");

	/* deallocating memory. */
	free(first);
	free(second);
	free(output);

	hipFree(dev_first);	CUDA_ASSERT_ERROR();
	hipFree(dev_second);	CUDA_ASSERT_ERROR();
	hipFree(dev_output);	CUDA_ASSERT_ERROR();

	return (0);
}