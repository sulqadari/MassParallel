#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "picture/bmp_image.h"

#define GET_MS(start, stop)					\
	(stop.tv_sec - start.tv_sec) * 1000.0 +	\
	(stop.tv_usec - start.tv_usec) / 1000.0

static void
usage(void)
{
	fprintf(stderr, "ERROR\nUsage: ./%s "
	"<path/to/input_image.bmp> "
	"<path/to/output_image.bmp>\n", EXECUTABLE_NAME);
	exit(1);
}

int
main(int argc, char*argv[])
{
	struct timeval start, stop;
	double elapsed;
	BMP_image picture;
	BMP_Info* info;

	uint8_t* pixels_d;
	if (argc < 3)
		usage();
	
	if (bmp_load_file(&picture, argv[1]))
		return (1);

	bmp_init_image(&picture);
	bmp_print_info(&picture);

	info = &picture.info;

	do {
		hipMalloc(&pixels_d, picture.info.image_size);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		hipMemcpy(pixels_d, &picture.buff[picture.info.header.pixels_offset],
					picture.info.image_size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		gettimeofday(&start, NULL);

		dim3 grid_(ceil(picture.info.pic_width / 16.0), ceil(picture.info.pic_height / 16.0), 1);
		dim3 block_(16, 16, 1);

		bmp_color_to_gray<<<grid_, block_>>>(pixels_d,
											info->pic_width,
											info->pic_height,
											info->bit_count);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		gettimeofday(&stop, NULL);

		elapsed = GET_MS(start, stop);
		printf("elapsed time: %.02f ms.\n", elapsed);

		hipMemcpy(&picture.buff[picture.info.header.pixels_offset], pixels_d,
					picture.info.image_size, hipMemcpyDeviceToHost);
		bmp_save_file(&picture, argv[2]);
	} while (0);

	hipFree(pixels_d);

	bmp_free(&picture);

	return (0);
}