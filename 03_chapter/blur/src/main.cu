#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "picture/bmp_image.h"

#define GET_MS(start, stop)					\
	(stop.tv_sec - start.tv_sec) * 1000.0 +	\
	(stop.tv_usec - start.tv_usec) / 1000.0

static void
usage(void)
{
	fprintf(stderr, "ERROR\nUsage: ./%s "
	"<path/to/input_image.bmp> "
	"<path/to/output_image.bmp>\n", EXECUTABLE_NAME);
	exit(1);
}

int
main(int argc, char*argv[])
{
	struct timeval start, stop;
	double elapsed;
	BMP_image picture;
	BMP_Info* info;

	uint8_t* inputImg;
	uint8_t* outputImg;

	if (argc < 3)
		usage();
	
	if (bmp_load_file(&picture, argv[1]))
		return (1);

	bmp_init_image(&picture);
	bmp_print_info(&picture);

	info = &picture.info;

	do {
		hipMalloc(&inputImg, picture.info.image_size);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		hipMemcpy(inputImg, &picture.buff[picture.info.header.pixels_offset],
							picture.info.image_size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		hipMalloc(&outputImg, picture.info.image_size);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		gettimeofday(&start, NULL);

		dim3 grid_(ceil(picture.info.pic_width / 16.0), ceil(picture.info.pic_height / 16.0), 1);
		dim3 block_(16, 16, 1);

		bmp_blurring<<<grid_, block_>>>(outputImg, inputImg,
											info->pic_width,
											info->pic_height,
											1);
		hipDeviceSynchronize();
		CUDA_ASSERT_ERROR(hipGetLastError());

		gettimeofday(&stop, NULL);

		elapsed = GET_MS(start, stop);
		printf("elapsed time: %.02f ms.\n", elapsed);

		hipMemcpy(&picture.buff[picture.info.header.pixels_offset], outputImg,
					picture.info.image_size, hipMemcpyDeviceToHost);
		bmp_save_file(&picture, argv[2]);
	} while (0);

	hipFree(inputImg);
	hipFree(outputImg);

	bmp_free(&picture);

	return (0);
}